#include "hip/hip_runtime.h"
#include <cstdlib>
#include <ctime>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include "functions.h"

typedef unsigned long size_n;

struct Attributes {
    int rows;
    int cols;
    size_n N;
    size_n l_x;
    size_n l_y;
    double prob_crossing;
    double prob_mutation;

    size_n array_size;

    size_n get_array_size() {
        array_size = N * (l_x + l_y);
        return array_size;
    }
};


double limit_x [] = {-5.12, 5.12};
double limit_y [] = {-5.12, 5.12};


__host__ __device__ double drop_wave(double x, double y){
    return -((1+cos(12*sqrt(pow(x, 2)+pow(y, 2))))/(0.5*pow(x, 2)+pow(y, 2)+2));
}

__constant__ Attributes attrs;

__host__ __device__ double get_real(
    bool * array,
    int length,
    double left,
    double right,
    size_n start,
    size_n end
){
    int sum = 0;
    int j = 0;
    for(size_n i = start; i < end; ++i, ++j) {
        // printf("%d:", array[i]);
        sum += array[i] * pow(2, length - j - 1);
    }
    // printf("\n");
    return left + sum * ((right - left) / (pow(2, length) - 1));
}

__device__ float generate( hiprandState* globalState, int ind )
{
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState;
    return RANDOM;
}

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init( seed, id, 0, &state[id] );
}


__device__ int chooseNeighbor (int idx, hiprandState* st, int cols, int rows, int c, int r) {
    int k = generate(st, idx) * 100000;
    k = (k / 100) % 4;
    size_n odx = 0;

    if (k == 0) {
        if ((c + 1) < cols) {
            odx = idx + 1;
        } else {
            odx = idx - cols + 1;
        }
    } else if (k == 1) {
        if ((c - 1) >= 0) {
            odx = idx - 1;
        } else {
            odx = idx + cols - 1;
        }
    } else if (k == 2) {
        if ((r + 1) < cols) {
            odx = idx + cols;
        } else {
            odx = idx % cols;
        }
    } else if (k == 3) {
        if ((r - 1) >= 0) {
            odx = idx - cols;
        } else {
            odx = (cols * (rows - 1)) + idx;
        }
    }
    return odx;
}


__global__ void tournament(bool* a, bool* o, hiprandState* st) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int cols = attrs.cols;
    int rows = attrs.rows;
    int r = idx / rows;
    int c = idx % cols;
    if (idx >= attrs.N) {
        return;
    }

    int odx = chooseNeighbor(idx, st, cols, rows, c, r);

    int ai = idx * (attrs.l_x + attrs.l_y);
    int oi = odx * (attrs.l_x + attrs.l_y);

    double ax = get_real(a, attrs.l_x, -5.12, 5.12, ai, ai + attrs.l_x);
    double ay = get_real(
        a, attrs.l_y, -5.12, 5.12, ai + attrs.l_x, ai + attrs.l_x + attrs.l_y);

    double ox = get_real(a, attrs.l_x, -5.12, 5.12, oi, oi + attrs.l_x);
    double oy = get_real(
        a, attrs.l_y, -5.12, 5.12, oi + attrs.l_x, oi + attrs.l_x + attrs.l_y);

    // printf("idx: %d (%f, %f) -> odx: %d (%f, %f)\n", idx, ax, ay, odx, ox, oy);
    double fa = drop_wave(ax, ay);
    double fo = drop_wave(ox, oy);

    int start;
    if (fa > fo) {
        start = ai;
    } else {
        start = oi;
    }
    for (int i = 0; i < (attrs.l_x + attrs.l_y); i++) {
        o[ai + i] = a[start + i];
    }
}

__global__ void reproduction(bool * a, bool * o, hiprandState* st) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int cols = attrs.cols;
    int rows = attrs.rows;
    int r = idx / rows;
    int c = idx % cols;
    if (idx >= attrs.N) {
        return;
    }

    double p_mutation = generate(st, idx);
    double p_crossing = generate(st, idx);

    if (p_mutation < attrs.prob_mutation) {
        int k = generate(st, idx) * (attrs.l_x + attrs.l_y) * 100;
        k = k % (attrs.l_x + attrs.l_y);
        int ai = idx * (attrs.l_x + attrs.l_y);
        o[ai + k] = !a[ai + k];
    }
    if (p_crossing < attrs.prob_crossing) {
        int k = generate(st, idx) * (attrs.l_x + attrs.l_y) * 100;
        k = k % (attrs.l_x + attrs.l_y);
        int odx = chooseNeighbor(idx, st, cols, rows, c, r);
        int ai = idx * (attrs.l_x + attrs.l_y);
        int ao = odx * (attrs.l_x + attrs.l_y);

        for (int i = 0; i < (attrs.l_x + attrs.l_y); i++) {
            if (i < k) {
                o[ai + i] = a[ao + i];
            } else {
                o[ai + i] = a[ai + i];
            }

        }
    }
}


void printMatrix(bool * & mat, Attributes & attrs, int limit = 10) {
    size_n inc = attrs.l_x + attrs.l_y;
    int i = 0;
    for (int x = 0; x < attrs.array_size; x += inc, i++) {
        if (i > limit) {
            break;
        }
        double realx = get_real(
            mat, attrs.l_x, limit_x[0], limit_x[1], x, x + attrs.l_x);
        double realy = get_real(
            mat, attrs.l_y, limit_y[0], limit_y[1], x + attrs.l_x, x + attrs.l_x + attrs.l_y);
        printf("%d: %f, %f\n", i, realx, realy);
    }
}


void random_bitvector(bool * & arr, size_n size) {
    for (size_n i = 0; i < size; i++) {
        int number = rand() % 2;
        arr[i] = number;
    }
}

void printBitVector(bool * & arr, size_n size, int limit = 10) {
    for (size_n i = 0; i < size; i++) {
        if (i > limit) break;
        printf("%d - ", arr[i]);
    }
}


int main () {
    srand((unsigned)time(0));
    int iterations = 100;

    size_n cols = 31;
    size_n rows = 31;
    size_n precission = 10;
    Attributes A;
    A.cols = cols;
    A.rows = rows;
    A.prob_crossing = 0.75;
    A.prob_mutation = 0.10;
    A.N = cols * rows;
    A.l_x = get_l(limit_x[0], limit_x[1], precission);
    A.l_y = get_l(limit_y[0], limit_y[1], precission);

    size_n arraySize = A.get_array_size();

    bool * a = new bool [arraySize];
    bool * x = new bool [arraySize];
    bool * da, * dx;

    size_n size = (arraySize) * sizeof(bool);

    hipMemcpyToSymbol(HIP_SYMBOL(attrs), &A, sizeof(Attributes));
    hipMalloc((void **) &da, size);
    hipMalloc((void **) &dx, size);

    hiprandState *d_state;

    hipMalloc(&d_state, A.N * sizeof(hiprandState));

    random_bitvector(a, arraySize);


    int maxThreads = (A.N > 128) ? 128 : A.N;
    int blocks = (A.N + maxThreads - 1) / maxThreads;

    printMatrix(a, A);

    hipMemcpy(da, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dx, x, size, hipMemcpyHostToDevice);

    setup_kernel <<< blocks, maxThreads >>> (d_state, unsigned(time(NULL)) );

    for (int i = 0; i < iterations; i++) {
        tournament <<< blocks, maxThreads >>> (da, dx, d_state);
        hipMemcpy(da, dx, size, hipMemcpyDeviceToDevice);
        reproduction <<< blocks, maxThreads >>> (da, dx, d_state);
        hipMemcpy(da, dx, size, hipMemcpyDeviceToDevice);
    }

    hipMemcpy(a, da, size, hipMemcpyDeviceToHost);

    printf("\n\n");
    printMatrix(a, A);

    printf("%f\n", drop_wave(-5.120000, -5.120000));

    return 0;
}
